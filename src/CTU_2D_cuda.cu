
#include <hip/hip_runtime.h>
/*! \file CTU_2D_cuda.cu
 *  \brief Definitions of the cuda 2D CTU algorithm functions. */

#ifdef CUDA

#include<stdio.h>
#include<math.h>
#include<cuda.h>
#include"global.h"
#include"global_cuda.h"
#include"hydro_cuda.h"
#include"CTU_2D_cuda.h"
#include"pcm_cuda.h"
#include"plmp_ctu_cuda.h"
#include"plmc_cuda.h"
#include"ppmp_ctu_cuda.h"
#include"ppmc_cuda.h"
#include"exact_cuda.h"
#include"roe_cuda.h"
#include"hllc_cuda.h"
#include"h_correction_2D_cuda.h"
#include"cooling_cuda.h"
#include"subgrid_routines_2D.h"



__global__ void Evolve_Interface_States_2D(Real *dev_Q_Lx, Real *dev_Q_Rx, Real *dev_F1_x,
                                           Real *dev_Q_Ly, Real *dev_Q_Ry, Real *dev_F1_y,
                                           int nx, int ny, int n_ghost, Real dx, Real dy, Real dt);


Real CTU_Algorithm_2D_CUDA(Real *host_conserved0, Real *host_conserved1, int nx, int ny, int x_off, int y_off, int n_ghost, Real dx, Real dy, Real xbound, Real ybound, Real dt)
{

  //Here, *host_conserved contains the entire
  //set of conserved variables on the grid
  //concatenated into a 1-d array
  //host_conserved0 contains the values at time n,
  //host_conserved1 will contain the values at time n+1

  #ifdef TIME
  // capture the start time
  cudaEvent_t start, stop;
  cudaEventCreate(&start);
  cudaEventCreate(&stop);
  float elapsedTime;
  #endif

  int n_fields = 5;
  #ifdef DE
  n_fields++;
  #endif


  // dimensions of subgrid blocks
  int nx_s, ny_s;
  int nz_s = 1;
  int x_off_s, y_off_s; // x and y offsets for subgrid block

  // total number of blocks needed
  int block_tot;    //total number of subgrid blocks (unsplit == 1)
  int block1_tot;   //total number of subgrid blocks in x direction
  int block2_tot;   //total number of subgrid blocks in y direction
  int remainder1;   //modulus of number of cells after block subdivision in x direction
  int remainder2;   //modulus of number of cells after block subdivision in y direction

  // counter for which block we're on
  int block = 0;

  // calculate the dimensions for each subgrid block
  sub_dimensions_2D(nx, ny, n_ghost, &nx_s, &ny_s, &block1_tot, &block2_tot, &remainder1, &remainder2, n_fields);
  block_tot = block1_tot*block2_tot;
  //printf("%d %d %d %d %d %d %d %d\n", nx, ny, nx_s, ny_s, block1_tot, block2_tot, remainder1, remainder2);

  // number of cells in one subgrid block
  int BLOCK_VOL = nx_s*ny_s*nz_s;

  // define the dimensions for the 2D grid
  //int  ngrid = (n_cells + TPB - 1) / TPB;
  int  ngrid = (BLOCK_VOL + 2*TPB - 1) / (2*TPB);

  //number of blocks per 2-d grid  
  dim3 dim2dGrid(ngrid, 2, 1);

  //number of threads per 1-d block   
  dim3 dim1dBlock(TPB, 1, 1);

  // Set up pointers for the location to copy from and to
  Real *tmp1;
  Real *tmp2;

  // allocate buffer to copy conserved variable blocks from and to 
  Real *buffer;
  if (block_tot > 1) {
    if ( NULL == ( buffer = (Real *) malloc(n_fields*BLOCK_VOL*sizeof(Real)) ) ) {
      printf("Failed to allocate CPU buffer.\n");
    }
    tmp1 = buffer;
    tmp2 = buffer;
  }
  else {
    tmp1 = host_consereved0;
    tmp2 = host_consereved1;
  }  

  // allocate an array on the CPU to hold max_dti returned from each thread block
  Real max_dti = 0;
  Real *host_dti_array;
  host_dti_array = (Real *) malloc(2*ngrid*sizeof(Real));
  #ifdef COOLING_GPU
  Real min_dt = 1e10;
  Real *host_dt_array;
  host_dt_array = (Real *) malloc(2*ngrid*sizeof(Real));
  #endif  
  
  // allocate GPU arrays
  // conserved variables
  Real *dev_conserved;
  // input states and associated interface fluxes (Q* and F* from Stone, 2008)
  Real *Q_Lx, *Q_Rx, *Q_Ly, *Q_Ry, *F_x, *F_y;
  // arrays to hold the eta values for the H correction
  Real *eta_x, *eta_y, *etah_x, *etah_y;
  // array of inverse timesteps for dt calculation
  Real *dev_dti_array;
  #ifdef COOLING_GPU
  // array of timesteps for dt calculation (cooling restriction)
  Real *dev_dt_array;
  #endif

  // allocate memory on the GPU
  CudaSafeCall( cudaMalloc((void**)&dev_conserved, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Lx, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Rx, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Ly, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Ry, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F_x,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F_y,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&eta_x,   BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&eta_y,   BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah_x,  BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah_y,  BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&dev_dti_array, 2*ngrid*sizeof(Real)) );
  #ifdef COOLING_GPU
  CudaSafeCall( cudaMalloc((void**)&dev_dt_array, ngrid*sizeof(Real)) );
  #endif 


  // START LOOP OVER SUBGRID BLOCKS HERE
  while (block < block_tot) {

    // copy the conserved variable block to the buffer
    host_copy_block_2D(nx, ny, nx_s, ny_s, n_ghost, block, block1_tot, block2_tot, remainder1, remainder2, BLOCK_VOL, host_conserved0, buffer, n_fields);

    // calculate the global x and y offsets of this subgrid block
    // (only needed for gravitational potential)
    get_offsets_2D(nx_s, ny_s, n_ghost, x_off, y_off, block, block1_tot, block2_tot, remainder1, remainder2, &x_off_s, &y_off_s);    

/*
    // zero all the GPU arrays
    cudaMemset(dev_conserved, 0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(Q_Lx,  0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(Q_Rx,  0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(Q_Ly,  0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(Q_Ry,  0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(F_x,   0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(F_y,   0, n_fields*BLOCK_VOL*sizeof(Real));
    cudaMemset(eta_x,  0,  BLOCK_VOL*sizeof(Real));
    cudaMemset(eta_y,  0,  BLOCK_VOL*sizeof(Real));
    cudaMemset(etah_x, 0,  BLOCK_VOL*sizeof(Real));
    cudaMemset(etah_y, 0,  BLOCK_VOL*sizeof(Real));
    cudaMemset(dev_dti_array, 0, 2*ngrid*sizeof(Real));
    CudaCheckError();
*/

    // copy the conserved variables onto the GPU
    CudaSafeCall( cudaMemcpy(dev_conserved, tmp1, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyHostToDevice) );


    // Step 1: Do the reconstruction
    #ifdef PCM
    PCM_Reconstruction_2D<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, Q_Ly, Q_Ry, nx_s, ny_s, n_ghost, gama);
    #endif
    #ifdef PLMP
    PLMP_CTU<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0);
    PLMP_CTU<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1);
    #endif
    #ifdef PLMC
    PLMC_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0);
    PLMC_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1);
    #endif
    #ifdef PPMP
    PPMP_CTU<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0);
    PPMP_CTU<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1);
    #endif
    #ifdef PPMC
    PPMC_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0);
    PPMC_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1);
    #endif
    CudaCheckError();

    #ifdef H_CORRECTION
    #ifndef CTU
    // Step 3.5: Calculate eta values for H correction
    calc_eta_x_2D<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, eta_x, nx_s, ny_s, n_ghost, gama);
    calc_eta_y_2D<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, eta_y, nx_s, ny_s, n_ghost, gama);
    CudaCheckError();
    // and etah values for each interface
    calc_etah_x_2D<<<dim2dGrid,dim1dBlock>>>(eta_x, eta_y, etah_x, nx_s, ny_s, n_ghost);
    calc_etah_y_2D<<<dim2dGrid,dim1dBlock>>>(eta_x, eta_y, etah_y, nx_s, ny_s, n_ghost);
    CudaCheckError();
    #endif // NO CTU
    #endif // H_CORRECTION


    // Step 2: Calculate the fluxes
    #ifdef EXACT
    Calculate_Exact_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0);
    Calculate_Exact_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1);
    #endif
    #ifdef ROE
    Calculate_Roe_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0);
    Calculate_Roe_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1);
    #endif
    #ifdef HLLC 
    Calculate_HLLC_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0);
    Calculate_HLLC_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1);
    #endif
    CudaCheckError();

#ifdef CTU

    // Step 3: Evolve the interface states
    Evolve_Interface_States_2D<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, Q_Ly, Q_Ry, F_y, nx_s, ny_s, n_ghost, dx, dy, dt);
    CudaCheckError();


    #ifdef H_CORRECTION
    // Step 3.5: Calculate eta values for H correction
    calc_eta_x_2D<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, eta_x, nx_s, ny_s, n_ghost, gama);
    calc_eta_y_2D<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, eta_y, nx_s, ny_s, n_ghost, gama);
    CudaCheckError();
    // and etah values for each interface
    calc_etah_x_2D<<<dim2dGrid,dim1dBlock>>>(eta_x, eta_y, etah_x, nx_s, ny_s, n_ghost);
    calc_etah_y_2D<<<dim2dGrid,dim1dBlock>>>(eta_x, eta_y, etah_y, nx_s, ny_s, n_ghost);
    CudaCheckError();
    #endif


    // Step 4: Calculate the fluxes again
    #ifdef EXACT
    Calculate_Exact_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0);
    Calculate_Exact_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1);
    #endif
    #ifdef ROE
    Calculate_Roe_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0);
    Calculate_Roe_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1);
    #endif
    #ifdef HLLC
    Calculate_HLLC_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0);
    Calculate_HLLC_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1);
    #endif
    CudaCheckError();

#endif //CTU


    // Step 5: Update the conserved variable array
    Update_Conserved_Variables_2D<<<dim2dGrid,dim1dBlock>>>(dev_conserved, F_x, F_y, nx_s, ny_s, x_off_s, y_off_s, n_ghost, dx, dy, xbound, ybound, dt, gama);
    CudaCheckError();

    // Synchronize the total and internal energy
    #ifdef DE
    Sync_Energies_2D<<<dim2dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, n_ghost, gama);
    CudaCheckError();    
    #endif

    // Apply cooling
    #ifdef COOLING_GPU
    printf("Need to fix cooling.\n");
    //cooling_kernel<<<dim2dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, nz_s, n_ghost, dt, gama);
    //CudaCheckError();    
    #endif

    // Step 6: Calculate the next timestep
    Calc_dt_2D<<<dim2dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, n_ghost, dx, dy, dev_dti_array, gama);
    CudaCheckError();    


    // copy the conserved variable array back to the CPU
    CudaSafeCall( cudaMemcpy(tmp2, dev_conserved, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyDeviceToHost) );

    // copy the updated conserved variable array back into the host_conserved array on the CPU
    host_return_block_2D(nx, ny, nx_s, ny_s, n_ghost, block, block1_tot, block2_tot, remainder1, remainder2, BLOCK_VOL, host_conserved1, buffer, n_fields);


    // copy the dti array onto the CPU
    CudaSafeCall( cudaMemcpy(host_dti_array, dev_dti_array, 2*ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
    // iterate through to find the maximum inverse dt for this subgrid block
    for (int i=0; i<2*ngrid; i++) {
      max_dti = fmax(max_dti, host_dti_array[i]);
    }
    #ifdef COOLING_GPU
    // copy the dt array from cooling onto the CPU
    CudaSafeCall( cudaMemcpy(host_dt_array, dev_dt_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
    // iterate through to find the minimum dt for this subgrid block
    for (int i=0; i<2*ngrid; i++) {
      min_dt = fmin(min_dt, host_dt_array[i]);
    }  
    //printf("%f %f\n", min_dt, 0.3/max_dti); 
    if (min_dt < 0.3/max_dti) {
      //printf("%f %f\n", min_dt, 0.3/max_dti); 
      min_dt = fmax(min_dt, 1.0);
      max_dti = 0.3/min_dt;
    }
    #endif


    // add one to the counter
    block++;

  }


  // free the CPU memory
  free(host_dti_array);
  if (block_tot > 1) free(buffer);
  #ifdef COOLING_GPU
  free(host_dt_array);  
  #endif    

  // free the GPU memory
  cudaFree(dev_conserved);
  cudaFree(Q_Lx);
  cudaFree(Q_Rx);
  cudaFree(Q_Ly);
  cudaFree(Q_Ry);
  cudaFree(F_x);
  cudaFree(F_y);
  cudaFree(eta_x);
  cudaFree(eta_y);
  cudaFree(etah_x);
  cudaFree(etah_y);
  cudaFree(dev_dti_array);
  #ifdef COOLING_GPU
  cudaFree(dev_dt_array);
  #endif

  // return the maximum inverse timestep
  return max_dti;

}





__global__ void Evolve_Interface_States_2D(Real *dev_Q_Lx, Real *dev_Q_Rx, Real *dev_F_x, 
                                           Real *dev_Q_Ly, Real *dev_Q_Ry, Real *dev_F_y,
                                           int nx, int ny, int n_ghost, Real dx, Real dy, Real dt)
{
  Real dtodx = dt/dx;
  Real dtody = dt/dy;
  int n_cells = nx*ny;

  // get a thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  int tid = threadIdx.x + blockId * blockDim.x;
  int yid = tid / nx;
  int xid = tid - yid*nx;
  int id = xid + yid*nx;


  // set the new x interface states
  if (xid > n_ghost-2 && xid < nx-n_ghost && yid > n_ghost-2 && yid < ny-n_ghost+1)
  {
    // left
    int ipo = xid+1 + yid*nx;
    int jmo = xid + (yid-1)*nx;
    int ipojmo = xid+1 + (yid-1)*nx;
    dev_Q_Lx[            id] += 0.5*dtody*(dev_F_y[            jmo] - dev_F_y[            id]);
    dev_Q_Lx[  n_cells + id] += 0.5*dtody*(dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id]);
    dev_Q_Lx[2*n_cells + id] += 0.5*dtody*(dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id]);
    dev_Q_Lx[3*n_cells + id] += 0.5*dtody*(dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id]);
    dev_Q_Lx[4*n_cells + id] += 0.5*dtody*(dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id]);
    // right
    dev_Q_Rx[            id] += 0.5*dtody*(dev_F_y[            ipojmo] - dev_F_y[            ipo]);
    dev_Q_Rx[  n_cells + id] += 0.5*dtody*(dev_F_y[  n_cells + ipojmo] - dev_F_y[  n_cells + ipo]);
    dev_Q_Rx[2*n_cells + id] += 0.5*dtody*(dev_F_y[2*n_cells + ipojmo] - dev_F_y[2*n_cells + ipo]);
    dev_Q_Rx[3*n_cells + id] += 0.5*dtody*(dev_F_y[3*n_cells + ipojmo] - dev_F_y[3*n_cells + ipo]);
    dev_Q_Rx[4*n_cells + id] += 0.5*dtody*(dev_F_y[4*n_cells + ipojmo] - dev_F_y[4*n_cells + ipo]);
  }
  // set the new y interface states
  if (yid > n_ghost-2 && yid < ny-n_ghost && xid > n_ghost-2 && xid < nx-n_ghost+1)
  {
    // left
    int jpo = xid + (yid+1)*nx;
    int imo = xid-1 + yid*nx;
    int jpoimo = xid-1 + (yid+1)*nx;
    dev_Q_Ly[            id] += 0.5*dtodx*(dev_F_x[            imo] - dev_F_x[            id]); 
    dev_Q_Ly[  n_cells + id] += 0.5*dtodx*(dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id]); 
    dev_Q_Ly[2*n_cells + id] += 0.5*dtodx*(dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id]); 
    dev_Q_Ly[3*n_cells + id] += 0.5*dtodx*(dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id]); 
    dev_Q_Ly[4*n_cells + id] += 0.5*dtodx*(dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id]); 
    // right
    dev_Q_Ry[            id] += 0.5*dtodx*(dev_F_x[            jpoimo] - dev_F_x[            jpo]); 
    dev_Q_Ry[  n_cells + id] += 0.5*dtodx*(dev_F_x[  n_cells + jpoimo] - dev_F_x[  n_cells + jpo]); 
    dev_Q_Ry[2*n_cells + id] += 0.5*dtodx*(dev_F_x[2*n_cells + jpoimo] - dev_F_x[2*n_cells + jpo]); 
    dev_Q_Ry[3*n_cells + id] += 0.5*dtodx*(dev_F_x[3*n_cells + jpoimo] - dev_F_x[3*n_cells + jpo]); 
    dev_Q_Ry[4*n_cells + id] += 0.5*dtodx*(dev_F_x[4*n_cells + jpoimo] - dev_F_x[4*n_cells + jpo]); 
  }

}


#endif //CUDA

